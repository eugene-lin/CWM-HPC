
#include <stdlib.h>
#include <stdio.h>
//
// these libraries are for CUDA RNG
#include <hip/hip_runtime.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>

// define number of cores to use
#define N 20

// define number of trials to use
#define M 10000000

// we will be using unsigned ints because we want a random positive number between 0 and 1
// kernel for generating random number then executing monte carlo method
__global__ void mcarlo( float *d_area, int totarea ) {
	
	int i;
	totarea = 0;
	unsigned int seed = threadIdx.x;
	hiprandState s;
	
	// seed a random number generator
	hiprand_init(seed, 0, 0, &s);

	// for loop to implement monte carlo
	for(i = 0; i < M; ++i) {
		int ind = blockIdx.x * blockDim.x + threadIdx.x;
		float x = hiprand_uniform(&s);
		float y = hiprand_uniform(&s);
		if( x*x + y*y <= 1.0f) {
			d_area[ind] += 1;
		}		
	}

	// sum up elements in array
	i = 0;
	while( i < M )
        {
                totarea += d_area[i];
                i++;
        }

}

// main function
int main( void ) {
	// initiate host variables
	int h_tot;
	int totarea = 0;

	//allocate host memory (unnecessary?)
	//h_area = (float*)  malloc(M * sizeof(float));

	// initiate device
	int deviceid = 0;
	int devCount;
	hipGetDeviceCount(&devCount);
	if(deviceid<devCount) hipSetDevice(deviceid);
	else return(1);

	// define and allocate memory on the device
	float *d_area;
	hipMalloc(&d_area, M*sizeof(float));	

	// execute kernel to implement mcarlo
	mcarlo<<<N , 1>>>( d_area, totarea );

	// transfer result back to host
	hipMemcpy(&h_tot, &totarea, sizeof(int), hipMemcpyDeviceToHost);
	
	// calculate pi
	int PI = 4 * h_tot / N;

	// display result
	printf("\nPI is %f\n", PI);

	// free memory 
	hipFree(d_area);
	
	return 0;
}
